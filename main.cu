#include <iostream>
#include <hip/hip_runtime.h>

// Kernel para somar dois vetores
__global__ void somaVetores(int *a, int *b, int *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 100;
    int size = n * sizeof(int);

    // Aloca memória no host
    int *h_a = (int *)malloc(size);
    int *h_b = (int *)malloc(size);
    int *h_c = (int *)malloc(size);

    // Inicializa os vetores a e b no host
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Aloca memória no device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copia os dados do host para o device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define o número de threads e blocos
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Executa o kernel
    somaVetores<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copia o resultado do device para o host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Exibe o resultado
    std::cout << "Resultado da soma:\n";
    for (int i = 0; i < n; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << "\n";

    // Libera a memória
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
